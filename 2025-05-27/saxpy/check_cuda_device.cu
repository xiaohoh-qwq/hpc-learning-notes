#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int count;
    hipError_t err = hipGetDeviceCount(&count);
    if (err != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }
    std::cout << "CUDA device count: " << count << std::endl;
    return 0;
}
