
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

int main() {
    int n = 1 << 20;
    float *x, *y, *d_x, *d_y;
    float a = 2.0f;

    x = new float[n]; y = new float[n];
    for (int i = 0; i < n; ++i) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    saxpy<<<(n + 255)/256, 256>>>(n, a, d_x, d_y);

    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "y[0] = " << y[0] << std::endl;

    hipFree(d_x); hipFree(d_y);
    delete[] x; delete[] y;
    return 0;
}
