#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(err) \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        return -1; \
    }

#define CHECK_CUBLAS(err) \
    if (err != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS error at " << __FILE__ << ":" << __LINE__ << std::endl; \
        return -1; \
    }

int main() {
    const int N = 1024;
    const float alpha = 2.0f;
    std::vector<float> x(N, 1.0f);
    std::vector<float> y(N, 2.0f);

    // ✅ 检查 GPU 是否可用
    int device_count = 0;
    hipError_t devErr = hipGetDeviceCount(&device_count);
    if (devErr != hipSuccess || device_count == 0) {
        std::cerr << "No CUDA-capable device is detected." << std::endl;
        return -1;
    }

    // ✅ 设置设备
    CHECK_CUDA(hipSetDevice(0));

    // 分配 GPU 内存
    float *d_x, *d_y;
    CHECK_CUDA(hipMalloc((void**)&d_x, N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_y, N * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_x, x.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_y, y.data(), N * sizeof(float), hipMemcpyHostToDevice));

    // 使用 cuBLAS 执行 saxpy
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));
    CHECK_CUBLAS(hipblasSaxpy(handle, N, &alpha, d_x, 1, d_y, 1));

    CHECK_CUBLAS(hipblasDestroy(handle));

    // 拷回主机并打印一个值以验证
    CHECK_CUDA(hipMemcpy(y.data(), d_y, N * sizeof(float), hipMemcpyDeviceToHost));
    std::cout << "y[0] = " << y[0] << std::endl;  // 应该是 2 + 2 * 1 = 4

    // 释放 GPU 内存
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));

    return 0;
}
